
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);


        hipEventRecord(start);
        // Do Something Here
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("Time elapsed: %f\n", milliseconds);
}
